#include"channel.h"

void kernelCheck( hipError_t error, domain_t domain, const char* function)
{


	error= hipGetLastError();			
		if(error !=hipSuccess)
		{
			const char* error_string= hipGetErrorString(error);
			printf("\n error  %s : %s domain.rank=%d \n", function, error_string,domain.rank);
			exit(1);
		}

	return;
}

extern void cufftCheck( hipfftResult error, domain_t domain,  const char* function )
{
	if(error != HIPFFT_SUCCESS)
	{
		printf("\n error  %s : %d domain.rank=%d \n", function, error,domain.rank);
		exit(1);
	}
		
	return;
}  
#ifdef USE_CUSPARSE
extern void cusparseCheck( hipsparseStatus_t error, domain_t domain,  const char* function )
{
	if(error != HIPSPARSE_STATUS_SUCCESS)
	{
		printf("\n error  %s : %d domain.rank=%d \n", function, error,domain.rank);
		exit(1);
	}
		
	return;
}  
#endif
extern void cublasCheck(hipblasStatus_t error, domain_t domain, const char* function )
{
	if(error !=  HIPBLAS_STATUS_SUCCESS)
	{
		printf("\n error  %s : %d domain.rank=%d \n", function, error,domain.rank);
		exit(1);
	}
		
	return;
}  


extern void cudaCheck( hipError_t error, domain_t domain, const char* function)
{
	if(error !=hipSuccess)
	{
		const char* error_string= hipGetErrorString(error);
		printf("\n error  %s : %s domain.rank=%d \n", function, error_string,domain.rank);
		exit(1);
	}
		

	return;
}



extern void mpiCheck( int error, const char* function)
{
	if(error !=0)
	{
		//printf("\n error_MPI %s \n",(char*)function);
		printf("error_mpi");		
		exit(1);
	}
		
	

	return;
}



